#include "hip/hip_runtime.h"
#include "DataCompute.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void xMul(const float* A, const float* B, float* out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < 1)
    {
        out[idx] = A[idx] * B[idx];
        printf("GPU: A = %f, B = %f, out = %f\n", A[idx], B[idx], out[idx]);
    }
}


DataCompute::DataCompute()
{
    (this->A) = (float*)malloc(sizeof(float));
    *(this->A) = 5.f;

    (this->B) = (float*)malloc(sizeof(float));
    *(this->B) = 7.5;

    (this->C) = (float*)malloc(sizeof(float));
    *(this->C) = 0.0;
}

DataCompute::~DataCompute()
{
    free(this->A);
    free(this->B);
    free(this->C);
}


void DataCompute::compute(float* A, float* B, float* C)
{
    printf("check\n");
    float *A_in = nullptr, *B_in = nullptr, *C_out = nullptr;
    hipMalloc(&A_in, sizeof(float));
    hipMalloc(&B_in, sizeof(float));
    hipMalloc(&C_out, sizeof(float));
    printf("check\n");

    hipMemcpy(A_in, A, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_in, B, sizeof(float), hipMemcpyHostToDevice);

    printf("check\n");
    int threads = 256;
    int blocks = 1;
    xMul<<<blocks, threads>>>(A_in, B_in, C_out);
    hipDeviceSynchronize();

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess)
        std::cerr << "Kernel error: " << hipGetErrorString(err) << std::endl;

    err = hipMemcpy(C, C_out, sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        std::cerr << "Memcpy error: " << hipGetErrorString(err) << std::endl;

    printf("check\n");
    hipMemcpy(C, C_out, sizeof(float), hipMemcpyDeviceToHost);
    
    printf("check C: %f\n", *C);
    hipFree(A_in);
    hipFree(B_in);
    hipFree(C_out);
    printf("check C: %f\n", *C);
}

void DataCompute::printC()
{
    printf("checking A: %f\n", *A);
    printf("checking B: %f\n", *B);
    printf("checking C: %f\n", *C);
    this->compute(this->A, this->B, this->C);
    std::string str = "NO GO";

    if ((*(this->C) == (7.5 * 5)))
        str = "GO";

    std::cout << "C is " << str << " for value of " << *(this->C) << std::endl;
}