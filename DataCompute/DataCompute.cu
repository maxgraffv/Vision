#include "hip/hip_runtime.h"
#include "DataCompute.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void xMul(const float* A, const float* B, float* out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < 1)
    {
        out[idx] = A[idx] * B[idx];
        // printf("GPU: A = %f, B = %f, out = %f\n", A[idx], B[idx], out[idx]);
    }
}

__global__ void conv2d(const float* img, const float* ker, float* out, int w, int h, int kw, int kh)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= w || y >= h)
        return;

    if(x == 0 && y == 0)
    for(int i = 0; i < 9; i++)
        printf("K= %f\n",img[i]);

    int kCenterX = kw / 2;
    int kCenterY = kh / 2;

    for (int c = 0; c < 3; ++c) { // kanały: 0=R,1=G,2=B

        float sum = 0.0f;

        for (int ky = 0; ky < kh; ++ky) {
            for (int kx = 0; kx < kw; ++kx) {
                int ix = x + kx - kCenterX;
                int iy = y + ky - kCenterY;

                float val = 0.0f;
                if (ix >= 0 && ix < w && iy >= 0 && iy < h) {
                    int pixelIndex = (iy * w + ix) * 3;
                    val = img[pixelIndex + c];
                }

                float k = ker[ky * kw + kx];
                sum += val * k;
            }
        }

        out[(y * w + x) * 3 + c] = sum;
    }
}


DataCompute::DataCompute()
{
    (this->A) = (float*)malloc(sizeof(float));
    *(this->A) = 5.f;

    (this->B) = (float*)malloc(sizeof(float));
    *(this->B) = 7.5;

    (this->C) = (float*)malloc(sizeof(float));
    *(this->C) = 0.0;
}

DataCompute::~DataCompute()
{
    free(this->A);
    free(this->B);
    free(this->C);
}


void DataCompute::compute(float* A, float* B, float* C)
{
    float *A_in = nullptr, *B_in = nullptr, *C_out = nullptr;
    hipMalloc(&A_in, sizeof(float));
    hipMalloc(&B_in, sizeof(float));
    hipMalloc(&C_out, sizeof(float));

    hipMemcpy(A_in, A, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_in, B, sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = 1;
    xMul<<<blocks, threads>>>(A_in, B_in, C_out);
    hipDeviceSynchronize();

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess)
        std::cerr << "Kernel error: " << hipGetErrorString(err) << std::endl;

    err = hipMemcpy(C, C_out, sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        std::cerr << "Memcpy error: " << hipGetErrorString(err) << std::endl;

    hipMemcpy(C, C_out, sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(A_in);
    hipFree(B_in);
    hipFree(C_out);
}

void DataCompute::printC()
{
    this->compute(this->A, this->B, this->C);
    std::string str = "NO GO";

    if ((*(this->C) == (7.5 * 5)))
        str = "GO";

    std::cout << "CUDA GPU is " << str << std::endl;
}

void DataCompute::applyKernel(QtImage& img, Matrix<float>& kernel)
{
    int w = img.getWidth();
    int h = img.getHeight();
    int kw = kernel.getWidth();
    int kh = kernel.getHeight();

    dim3 block(16, 16);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);

    float output[img.buffer().size()];
    std::fill(output, output + img.buffer().size(), 0.0f);

    float *devKer = nullptr, *devImg = nullptr, *devOut = nullptr;

    std::cout << kernel.buffer().size() << std::endl;
    
    hipMalloc(&devImg, sizeof(float)*img.buffer().size());
    hipMalloc(&devKer, sizeof(float)*kernel.buffer().size());
    hipMalloc(&devOut, sizeof(float)*img.buffer().size());

    hipMemcpy(devImg, img.buffer().data(), sizeof(float)* img.buffer().size(), hipMemcpyHostToDevice);
    hipMemcpy(devKer, kernel.buffer().data(), sizeof(float)*kernel.buffer().size(), hipMemcpyHostToDevice);

    conv2d<<<grid, block>>>(devImg, devKer, devOut, w, h, kw, kh);

    hipError_t err = hipDeviceSynchronize();
    if(err != hipSuccess)
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;


    err = hipMemcpy(output, devOut, sizeof(float)*img.buffer().size(), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        std::cerr << "Memcpy error: " << hipGetErrorString(err) << std::endl;

    hipFree(devImg);
    hipFree(devKer);
    hipFree(devOut);

    img.buffer() = std::vector<float>(output, output+(w*h*3));
}